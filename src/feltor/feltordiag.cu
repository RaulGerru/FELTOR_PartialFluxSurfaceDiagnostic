#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <string>
#include <functional>
#include "json/json.h"
#include "dg/algorithm.h"
#include "dg/geometries/geometries.h"
#include "dg/file/file.h"
using HVec = dg::HVec;
using DVec = dg::DVec;
using DMatrix = dg::DMatrix;
using IDMatrix = dg::IDMatrix;
using IHMatrix = dg::IHMatrix;
using Geometry = dg::CylindricalGrid3d;
#define MPI_OUT
#include "feltordiag.h"
    
    //GENERAL COMMENT RAUL: Everything NEW can be found with the searcher writting or "NEW",
    // or "conv" for the convolution (as it was defined up to this point) 
    //or "part_" for the partial flux surface average, that has a new definition, 
    //as it is described in the feltor.pdf document.
    
    
int main( int argc, char* argv[])
{
    if( argc < 3)
    {
        std::cerr << "Usage: "<<argv[0]<<" [input0.nc ... inputN.nc] [output.nc]\n";
        return -1;
    }
    for( int i=1; i<argc-1; i++)
        std::cout << argv[i]<< " ";
    std::cout << " -> "<<argv[argc-1]<<std::endl;

    //------------------------open input nc file--------------------------------//
    file::NC_Error_Handle err;
    int ncid_in;
    err = nc_open( argv[1], NC_NOWRITE, &ncid_in); //open 3d file
    size_t length;
    err = nc_inq_attlen( ncid_in, NC_GLOBAL, "inputfile", &length);
    std::string inputfile(length, 'x');
    err = nc_get_att_text( ncid_in, NC_GLOBAL, "inputfile", &inputfile[0]);
    err = nc_inq_attlen( ncid_in, NC_GLOBAL, "geomfile", &length);
    std::string geomfile(length, 'x');
    err = nc_get_att_text( ncid_in, NC_GLOBAL, "geomfile", &geomfile[0]);
    err = nc_close( ncid_in);
    Json::Value js,gs;
    file::string2Json(inputfile, js, file::comments::are_forbidden);
    file::string2Json(geomfile, gs, file::comments::are_forbidden);
    //we only need some parameters from p, not all
    const feltor::Parameters p(js, file::error::is_warning);
    const dg::geo::solovev::Parameters gp(gs);
    p.display();
    gp.display();
    std::vector<std::string> names_input{
        "electrons", "ions", "Ue", "Ui", "potential", "induction"
    };
 
    //-----------------Create Netcdf output file with attributes----------//
    int ncid_out;
    err = nc_create(argv[argc-1],NC_NETCDF4|NC_CLOBBER, &ncid_out);

    /// Set global attributes
    std::map<std::string, std::string> att;
    att["title"] = "Output file of feltor/src/feltor/feltordiag.cu";
    att["Conventions"] = "CF-1.7";
    ///Get local time and begin file history
    auto ttt = std::time(nullptr);
    auto tm = *std::localtime(&ttt);
    std::ostringstream oss;
    ///time string  + program-name + args
    oss << std::put_time(&tm, "%Y-%m-%d %H:%M:%S");
    for( int i=0; i<argc; i++) oss << " "<<argv[i];
    att["history"] = oss.str();
    att["comment"] = "Find more info in feltor/src/feltor.tex";
    att["source"] = "FELTOR";
    att["references"] = "https://github.com/feltor-dev/feltor";
    att["inputfile"] = inputfile;
    att["geomfile"] = geomfile;
    for( auto pair : att)
        err = nc_put_att_text( ncid_out, NC_GLOBAL,
            pair.first.data(), pair.second.size(), pair.second.data());

    //-------------------Construct grids-------------------------------------//
  
    const double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    const double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    const double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    const double Zmax=p.boxscaleZp*gp.a*gp.elongation;
     
    double eta_0=3*M_PI/2; //NEW: Defining center of partial fsa
    double eta_range=90.; //NEW: Defining the poloidal range of partial fsa

    dg::Grid2d   g2d_out( Rmin,Rmax, Zmin,Zmax,
        p.n_out, p.Nx_out, p.Ny_out, p.bcxN, p.bcyN);

    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);
    double RO=mag.R0(), ZO=0.;
    dg::geo::findOpoint( mag.get_psip(), RO, ZO);
    const double psipO = mag.psip()( RO, ZO);
    if( p.damping_alpha > 0.) 
    {
        double damping_psi0 = (1.-p.damping_boundary*p.damping_boundary)*psipO;
        double damping_alpha = -(2.*p.damping_boundary+p.damping_alpha)*p.damping_alpha*psipO;
        mag = dg::geo::createModifiedSolovevField(gp, damping_psi0+damping_alpha/2.,
                fabs(p.damping_alpha/2.), ((psipO>0)-(psipO<0)));
    } 
    dg::HVec psipog2d = dg::evaluate( mag.psip(), g2d_out);
    // Construct weights and temporaries

    dg::HVec transferH2d = dg::evaluate(dg::zero,g2d_out);
    dg::HVec part_transferH2dX; //NEW for partial fsa
    dg::HVec t2d_mp = dg::evaluate(dg::zero,g2d_out);
    dg::HVec t2d_pol_conv = dg::evaluate(dg::zero,g2d_out); //NEW for convolution
   
       
    ///---------------  Construct X-point grid ---------------------//
         
       
    //std::cout << "Type X-point grid resolution (n(3), Npsi(32), Neta(640)) Must be divisible by 8\n";
    std::cout << "Using default X-point grid resolution (n(3), Npsi(64), Neta(640))\n";
    unsigned npsi = 3, Npsi = 64, Neta =40;//set number of psivalues (NPsi % 8 == 0)
    //std::cin >> npsi >> Npsi >> Neta;
    std::cout << "You typed "<<npsi<<" x "<<Npsi<<" x "<<Neta<<"\n";
    std::cout << "Generate X-point flux-aligned grid!\n";
    double R_X = gp.R_0-1.1*gp.triangularity*gp.a;
    double Z_X = -1.1*gp.elongation*gp.a;
    dg::geo::findXpoint( mag.get_psip(), R_X, Z_X);
    dg::geo::CylindricalSymmTensorLvl1 monitor_chi = dg::geo::make_Xconst_monitor( mag.get_psip(), R_X, Z_X) ;
    dg::geo::SeparatrixOrthogonal generator(mag.get_psip(), monitor_chi, psipO, R_X, Z_X, mag.R0(), 0, 0, false);
    double fx_0 = 1./8.;
    double psipmax = dg::blas1::reduce( psipog2d, 0. ,thrust::maximum<double>()); //DEPENDS ON GRID RESOLUTION!!
    std::cout << "psi max is            "<<psipmax<<"\n";
    psipmax = -fx_0/(1.-fx_0)*psipO;
    std::cout << "psi max in g1d_out is "<<psipmax<<"\n";
    dg::geo::CurvilinearGridX2d gridX2d( generator, fx_0, 0., npsi, Npsi, Neta, dg::DIR_NEU, dg::NEU);
    std::cout << "psi max in gridX2d is "<<gridX2d.x1()<<"\n";
    std::cout << "psi min in gridX2d is "<<gridX2d.x0()<<"\n";
    std::cout << "eta max in gridX2d is "<<gridX2d.y1()<<"\n";
    std::cout << "eta min in gridX2d is "<<gridX2d.y0()<<"\n"; 
    std::cout << "DONE!\n"; 
    //Create 1d grid 
    dg::Grid1d g1d_out(psipO, psipmax, 3, Npsi, dg::DIR_NEU); //inner value is always 0
    dg::Grid1d g1d_out_eta(gridX2d.y0(), gridX2d.y1(), 3, Neta, dg::DIR_NEU); ////NEW 1D grid for the eta (poloidal) directions instead of psi
    const double f0 = ( gridX2d.x1() - gridX2d.x0() ) / ( psipmax - psipO );
    std::cout << "f0 is "<<f0<<"\n";
    dg::HVec t1d = dg::evaluate( dg::zero, g1d_out), fsa1d( t1d), part_fsa1d(t1d), conv1d=dg::evaluate( dg::zero, g1d_out_eta); //	NEW: Definition of partial fsa1d and convolution variables
    dg::HVec transfer1d = dg::evaluate(dg::zero,g1d_out);
           
    /// ------------------- Compute 1d flux labels ---------------------//
  
    std::vector<std::tuple<std::string, dg::HVec, std::string> > map1d;
    /// Compute flux volume label
    dg::Average<dg::HVec > poloidal_average( gridX2d.grid(), dg::coo2d::y);
    dg::Average<dg::HVec > radial_average( gridX2d.grid(), dg::coo2d::x); //NEW for the convolution to get the LCFS value
    dg::HVec dvdpsip, part_dvdpsip; //NEW partial fsa definitions
    //metric and map
    dg::SparseTensor<dg::HVec> metricX = gridX2d.metric();
    std::vector<dg::HVec > coordsX = gridX2d.map();
    dg::HVec volX2d = dg::tensor::volume2d( metricX); 
    dg::HVec transferH2dX(volX2d); //NEW definitions
    dg::HVec conv_transferH2dX=transferH2dX; //NEW definitions
    dg::HVec conv_def_transferH2dX;
    dg::blas1::pointwiseDot( coordsX[0], volX2d, volX2d); //R\sqrt{g}
    dg::HVec part_volX2d=volX2d, conv_volX2d; //NEW: DEFINE A PARTIAL VOLUME MATRIX TO APPLY THE CUT   
	double conv_window=7.5;
	dg::blas1::pointwiseDot(part_volX2d, dg::evaluate(dg::geo::Grid_cutter(eta_0, eta_range), gridX2d.grid()), part_volX2d); //cut the volume grid to do the partial flux surface integral                                              
    poloidal_average( volX2d,dvdpsip, false);  
    poloidal_average( part_volX2d, part_dvdpsip, false);    ////NEW poloidal average only in the cutted volume matrix for the volume vector.    
    dg::blas1::scal( dvdpsip, 4.*M_PI*M_PI*f0);
    dg::blas1::scal( part_dvdpsip, 4.*M_PI*M_PI*f0);	//NEW normalization for the volume vector
	std::cout << "Starting volume convolution \n";
	dg::geo::Convolution_def conv(conv_window, f0, gridX2d, Neta);
	conv_volX2d=conv.convoluted_grid();
    std::cout << "Volume convolution finished \n";
	  
    map1d.emplace_back( "dvdpsi", dvdpsip,
        "Derivative of flux volume with respect to flux label psi");
    map1d.emplace_back( "dvdpsi_part_at_"+std::to_string(eta_0)+"_"+std::to_string(eta_range), part_dvdpsip,
        "Partial derivative of flux volume with respect to flux label psi");
    dg::HVec X_psi_vol = dg::integrate(dvdpsip, g1d_out);
    map1d.emplace_back( "psi_vol", X_psi_vol,
        "Flux volume evaluated with X-point grid");

    /// Compute flux area label
    dg::HVec gradZetaX = metricX.value(0,0), X_psi_area;
    dg::blas1::transform( gradZetaX, gradZetaX, dg::SQRT<double>());
    dg::blas1::pointwiseDot( volX2d, gradZetaX, gradZetaX); //R\sqrt{g}|\nabla\zeta|
    poloidal_average( gradZetaX, X_psi_area, false);
    dg::blas1::scal( X_psi_area, 4.*M_PI*M_PI);
    map1d.emplace_back( "psi_area", X_psi_area,
        "Flux area evaluated with X-point grid");

    dg::HVec rho = dg::evaluate( dg::cooX1d, g1d_out);
    dg::blas1::axpby( -1./psipO, rho, +1., 1., rho); //transform psi to rho
    map1d.emplace_back("rho", rho,
        "Alternative flux label rho = 1-psi/psimin");
    dg::blas1::transform( rho, rho, dg::SQRT<double>());
    map1d.emplace_back("rho_p", rho,
        "Alternative flux label rho_p = sqrt(1-psi/psimin)"); 
    dg::geo::SafetyFactor qprof( mag);
    dg::HVec qprofile = dg::evaluate( qprof, g1d_out);
    map1d.emplace_back("q-profile", qprofile,
        "q-profile (Safety factor) using direct integration");
    map1d.emplace_back("psi_psi",    dg::evaluate( dg::cooX1d, g1d_out),
        "Poloidal flux label psi (same as coordinate)");
    dg::HVec psit = dg::integrate( qprofile, g1d_out);
    map1d.emplace_back("psit1d", psit,
        "Toroidal flux label psi_t integrated using q-profile");
    //we need to avoid integrating >=0 for total psi_t
    dg::Grid1d g1d_fine(psipO<0. ? psipO : 0., psipO<0. ? 0. : psipO, 3 ,Npsi,dg::DIR_NEU);
    qprofile = dg::evaluate( qprof, g1d_fine);
    dg::HVec w1d = dg::create::weights( g1d_fine);
    double psit_tot = dg::blas1::dot( w1d, qprofile);
    dg::blas1::scal ( psit, 1./psit_tot);
    dg::blas1::transform( psit, psit, dg::SQRT<double>());
    map1d.emplace_back("rho_t", psit,
        "Toroidal flux label rho_t = sqrt( psit/psit_tot)");

    // interpolate from 2d grid to X-point points
    dg::IHMatrix grid2gridX2d  = dg::create::interpolation(
        coordsX[0], coordsX[1], g2d_out);
    // interpolate fsa back to 2d or 3d grid
    dg::IHMatrix fsa2rzmatrix = dg::create::interpolation(
        psipog2d, g1d_out, dg::DIR_NEU);

    dg::HVec dvdpsip2d = dg::evaluate( dg::zero, g2d_out);
    dg::blas2::symv( fsa2rzmatrix, dvdpsip, dvdpsip2d);
    dg::HMatrix dpsi = dg::create::dx( g1d_out, dg::DIR_NEU);
 
    // define 2d and 1d and 0d dimensions and variables
    int dim_ids[3], tvarID;
    err = file::define_dimensions( ncid_out, dim_ids, &tvarID, g2d_out);
    //Write long description
    std::string long_name = "Time at which 2d fields are written";
    err = nc_put_att_text( ncid_out, tvarID, "long_name", long_name.size(),
            long_name.data());
          
    int dim_ids1d[2] = {dim_ids[0], 0}; //time,  psi
    err = file::define_dimension( ncid_out, &dim_ids1d[1], g1d_out, {"psi"} ); 
    std::map<std::string, int> id0d, id1d, id2d, id2dX;

    size_t count1d[2] = {1, g1d_out.n()*g1d_out.N()};
    size_t count1d_conv[2] = {1, gridX2d.n()*gridX2d.Ny()}; //NEW convolution variables for the plot (eta and eta, psi2)
    size_t count2d[3] = {1, g2d_out.n()*g2d_out.Ny(), g2d_out.n()*g2d_out.Nx()};
    size_t count2d_X[3]= {1, gridX2d.n()*gridX2d.Ny(), gridX2d.n()*gridX2d.Nx()};
    size_t start2d[3] = {0, 0, 0};
    
    
    int partial_dim_idsX[2]={0,0};
    err = file::define_dimensions(ncid_out, partial_dim_idsX,  gridX2d.grid(), {"eta", "psi2"});  //NEW definition of the names of the new coordinates
    int dim_idsX[3]={dim_ids[0], partial_dim_idsX[0], partial_dim_idsX[1]};
 
   
		long_name = "Flux surface label";
        err = nc_put_att_text( ncid_out, dim_idsX[1], "long_name",
            long_name.size(), long_name.data()); //NEW Description new coordinate
        long_name = "Flux angle";
        err = nc_put_att_text( ncid_out, dim_idsX[2], "long_name",
            long_name.size(), long_name.data()); //NEW Description new coordinate
        
                   
        err = nc_def_var( ncid_out, "xcc", NC_DOUBLE, 3, dim_idsX, &id2dX["xcc"]);
		long_name="Cartesian x-coordinate";
        err = nc_put_att_text( ncid_out, id2dX["xcc"], "long_name",
            long_name.size(), long_name.data()); //NEW variable for the transformation from X-grid to the Z, R grid
        err = nc_def_var( ncid_out, "ycc", NC_DOUBLE, 3, dim_idsX, &id2dX["ycc"]);
        long_name="Cartesian y-coordinate"; //NEW variable for the transformation from X-grid to the Z, R grid
        err = nc_put_att_text( ncid_out, id2dX["ycc"], "long_name",
            long_name.size(), long_name.data());
            
        err = nc_def_var( ncid_out, "volX2d_sqrt(g)", NC_DOUBLE, 3, dim_idsX, &id2dX["volX2d_sqrt(g)"]);
		long_name="Volume matrix_sqrt(g)";
        err = nc_put_att_text( ncid_out, id2dX["volX2d_sqrt(g)"], "long_name",
            long_name.size(), long_name.data());
            
         err = nc_def_var( ncid_out, "conv_volX2d_sqrt(g)", NC_DOUBLE, 3, dim_idsX, &id2dX["conv_volX2d_sqrt(g)"]);
		long_name="Convoluted_Volume matrix_sqrt(g)";
        err = nc_put_att_text( ncid_out, id2dX["conv_volX2d_sqrt(g)"], "long_name",
            long_name.size(), long_name.data());
            
            
       dg::HVec eta_c=dg::evaluate( dg::cooX1d, g1d_out_eta);
       std::string  name = "eta_coord"; //NEW variable to plot in the x axis the eta 1D coordinate
       err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 2, dim_idsX,
            &id2dX[name]);          
       long_name = "Poloidal coordinate";
		err = nc_put_att_text( ncid_out, id2dX[name], "long_name", long_name.size(),
            long_name.data());

    //write 1d static vectors (psi, q-profile, ...) into file
    for( auto tp : map1d)
    {
        int vid;
        err = nc_def_var( ncid_out, std::get<0>(tp).data(), NC_DOUBLE, 1,
            &dim_ids1d[1], &vid);
        err = nc_put_att_text( ncid_out, vid, "long_name",
            std::get<2>(tp).size(), std::get<2>(tp).data());
        err = nc_enddef( ncid_out);
        err = nc_put_var_double( ncid_out, vid, std::get<1>(tp).data());
        err = nc_redef(ncid_out);
    }

    for( auto& record : feltor::diagnostics2d_list)
    {
        std::string record_name = record.name;
        if( record_name[0] == 'j')
            record_name[1] = 'v';
        name = record_name + "_fluc2d";
        long_name = record.long_name + " (Fluctuations wrt fsa on phi = 0 plane.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 3, dim_ids,
            &id2d[name]);
        err = nc_put_att_text( ncid_out, id2d[name], "long_name", long_name.size(),
            long_name.data());

        name = record_name + "_fsa2d";
        long_name = record.long_name + " (Flux surface average interpolated to 2d plane.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 3, dim_ids,
            &id2d[name]);
        err = nc_put_att_text( ncid_out, id2d[name], "long_name", long_name.size(),
            long_name.data());
       
       
        if( record_name[0] == 'j'){ //NEW DEFINITION OF VARIABLE AS CONVOLUTION FOR CURRENTS (J's)
			name = record_name + "_conv2d"; //NEW Convoluted matrix in the X-grid
        long_name = record.long_name + " (2d values convoluted in 'small' angles.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 3, dim_idsX,
            &id2dX[name]);
        err = nc_put_att_text( ncid_out, id2dX[name], "long_name", long_name.size(),
            long_name.data()); 		
            
        name = record_name + "_Xgrid"; //NEW Original total data in the X-grid
        long_name = record.long_name + " (Toroidal average in the X grid)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 3, dim_idsX,
            &id2dX[name]);
        err = nc_put_att_text( ncid_out, id2dX[name], "long_name", long_name.size(),
            long_name.data()); 
            
            	name = record_name + "_conv1d_LCFS"; //NEW Variable for the 1D plot of the convolution
        long_name = record.long_name + " (1D Fluxes at LCFS, poloidal distribution)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 2, dim_idsX,
            &id2dX[name]);
        err = nc_put_att_text( ncid_out, id2dX[name], "long_name", long_name.size(),
            long_name.data()); 	
            
            			name = record_name + "_partX2d";// NEW THe cutted grid for the partial fsa in the X-Grid
        long_name = record.long_name + " (2d partial fsa in X_grid.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 3, dim_idsX,
            &id2dX[name]);
        err = nc_put_att_text( ncid_out, id2dX[name], "long_name", long_name.size(),
            long_name.data()); 	
          
		}
 
        name = record_name + "_fsa"; 
        long_name = record.long_name + " (Flux surface average.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 2, dim_ids1d,
            &id1d[name]);
        err = nc_put_att_text( ncid_out, id1d[name], "long_name", long_name.size(),
            long_name.data());
            
                name = record_name + "_part_fsa_at_"+std::to_string(eta_0)+"_"+std::to_string(eta_range); //NEW partial fsa variable
        long_name = record.long_name + " (Partial Flux surface average.)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 2, dim_ids1d,
            &id1d[name]);
        err = nc_put_att_text( ncid_out, id1d[name], "long_name", long_name.size(),
            long_name.data());

        name = record_name + "_ifs";
        long_name = record.long_name + " (wrt. vol integrated flux surface average)";
        if( record_name[0] == 'j')
            long_name = record.long_name + " (wrt. vol derivative of the flux surface average)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 2, dim_ids1d,
            &id1d[name]);
        err = nc_put_att_text( ncid_out, id1d[name], "long_name", long_name.size(),
            long_name.data());

        name = record_name + "_ifs_lcfs";
        long_name = record.long_name + " (wrt. vol integrated flux surface average evaluated on last closed flux surface)";
        if( record_name[0] == 'j')
            long_name = record.long_name + " (flux surface average evaluated on the last closed flux surface)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 1, dim_ids,
            &id0d[name]);
        err = nc_put_att_text( ncid_out, id0d[name], "long_name", long_name.size(),
            long_name.data());

        name = record_name + "_ifs_norm";
        long_name = record.long_name + " (wrt. vol integrated square flux surface average from 0 to lcfs)";
        if( record_name[0] == 'j')
            long_name = record.long_name + " (wrt. vol integrated square derivative of the flux surface average from 0 to lcfs)";
        err = nc_def_var( ncid_out, name.data(), NC_DOUBLE, 1, dim_ids,
            &id0d[name]);
        err = nc_put_att_text( ncid_out, id0d[name], "long_name", long_name.size(),
            long_name.data());
    } 
    /////////////////////////////////////////////////////////////////////////
    size_t counter = 0;
    int ncid;
    for( int j=1; j<argc-1; j++)
    {
        int timeID;

        size_t steps;
        std::cout << "Opening file "<<argv[j]<<"\n";
        try{
            err = nc_open( argv[j], NC_NOWRITE, &ncid); //open 3d file
        } catch ( file::NC_Error& error)
        {
            std::cerr << "An error occurded opening file "<<argv[j]<<"\n";
            std::cerr << error.what()<<std::endl;
            std::cerr << "Continue with next file\n";
            continue;
        }
        err = nc_inq_unlimdim( ncid, &timeID); //Attention: Finds first unlimited dim, which hopefully is time and not energy_time
        err = nc_inq_dimlen( ncid, timeID, &steps);
        //steps = 3;
        for( unsigned i=0; i<steps; i++)//timestepping
        {
            if( j > 1 && i == 0)
                continue; // else we duplicate the first timestep
            start2d[0] = i;
            size_t start2d_out[3] = {counter, 0,0};
            size_t startX2d_out[3] = {counter, 0,0}; //NEW For the time of the convoluted output
            size_t start1d_out[2] = {counter, 0};
            // read and write time
            double time=0.;
            err = nc_get_vara_double( ncid, timeID, start2d, count2d, &time);
            std::cout << counter << " Timestep = " << i <<"/"<<steps-1 << "  time = " << time << std::endl;
            counter++;
            err = nc_put_vara_double( ncid_out, tvarID, start2d_out, count2d, &time);
            err = nc_put_vara_double( ncid_out, tvarID, startX2d_out, count2d_X, &time);

             for( auto& record : feltor::diagnostics2d_list)
            {
                std::string record_name = record.name;
                if( record_name[0] == 'j')
                    record_name[1] = 'v';
                //1. Read toroidal average
                int dataID =0;
                bool available = true;
                try{
                    err = nc_inq_varid(ncid, (record.name+"_ta2d").data(), &dataID);
                } catch ( file::NC_Error& error)
                { 
                    if(  i == 0)
                    { 
                        std::cerr << error.what() <<std::endl;
                        std::cerr << "Offending variable is "<<record.name+"_ta2d\n";
                        std::cerr << "Writing zeros ... \n";
                    }
                    available = false;
                }
                if( available)
                {  
                    err = nc_get_vara_double( ncid, dataID,
                        start2d, count2d, transferH2d.data());
          
                    //2. Compute fsa, partial fsa and output fsa and partial fsa
                    dg::blas2::symv( grid2gridX2d, transferH2d, transferH2dX); //interpolate onto X-point grid
                    part_transferH2dX=transferH2dX; //NEW: DEFINE A TOTAL GRID FOR THE CUTTED VOLUME TO BE APPLIED 
                    conv_transferH2dX=transferH2dX; //NEW: DEFINE A TOTAL GRID FOR THE CUTTED VOLUME TO BE APPLIED 
                   
    
                    dg::blas1::pointwiseDot( transferH2dX, volX2d, transferH2dX); //multiply by sqrt(g)   
                    dg::blas1::pointwiseDot( part_transferH2dX, part_volX2d, part_transferH2dX); //NEW: multiply by sqrt(g) with the partial grid  
                    dg::HVec part_t1d=t1d; //NEW: DEFINE a new Partial 1d grid
                    poloidal_average( transferH2dX, t1d, false); //average over eta
                    poloidal_average( part_transferH2dX, part_t1d, false); //NEW: POloidal average in the partial grid
                    dg::blas1::scal( t1d, 4*M_PI*M_PI*f0); //
                    dg::blas1::scal( part_t1d, 4*M_PI*M_PI*f0); // NEW: As the average is done divided by 2 pi for the whole y axis of the grid, it is neccesary to multiply by the eta_range/PI.
                    dg::blas1::scal(part_t1d, 360/eta_range);
                    dg::blas1::pointwiseDivide( t1d, dvdpsip, fsa1d );
                    dg::blas1::pointwiseDivide( part_t1d, dvdpsip, part_fsa1d ); 
                    if( record_name[0] == 'j'){
                        dg::blas1::pointwiseDot( fsa1d, dvdpsip, fsa1d );
                        dg::blas1::pointwiseDot( part_fsa1d, dvdpsip, part_fsa1d ); 
                        dg::blas1::pointwiseDivide(transferH2dX, conv_volX2d, conv_transferH2dX);
                        
                        //conv_def_transferH2dX=dg::evaluate(dg::geo::convolution2(conv_transferH2dX, conv_window, f0, gridX2d), gridX2d.grid()); //NEW convolution variable
						//radial_average(conv_transferH2dX, conv1d, false); //NEW: Radial average to transform the convoluted matrix to the vector
						//dg::blas1::scal(conv1d, npsi*Npsi); //to make the radial average an average instead of an integral, as we are integrating all 0's that we don't want.
                           
					} 
                    //3. Interpolate fsa on 2d plane : <f>
                    dg::blas2::gemv(fsa2rzmatrix, fsa1d, transferH2d); //fsa on RZ grid //IT SHOULD BE WITHOUT X
                } 
                else
                {
                    dg::blas1::scal( fsa1d, 0.);
                    dg::blas1::scal( part_fsa1d, 0.);
                    dg::blas1::scal( transferH2d, 0.);
                } 
                err = nc_put_vara_double( ncid_out, id1d.at(record_name+"_fsa"),
                    start1d_out, count1d, fsa1d.data());
                err = nc_put_vara_double( ncid_out, id1d.at(record_name+"_part_fsa_at_"+std::to_string(eta_0)+"_"+std::to_string(eta_range)),
                    start1d_out, count1d, part_fsa1d.data());
                err = nc_put_vara_double( ncid_out, id2d.at(record_name+"_fsa2d"),
                    start2d_out, count2d, transferH2d.data() ); 
                    
                if( record_name[0] == 'j'){ //NEW DEFINITION OF VARIABLE AS CONVOLUTION FOR CURRENTS (J's)
				//err = nc_put_vara_double( ncid_out, id2dX.at(record_name+"_conv2d"),
                //startX2d_out, count2d_X, conv_transferH2dX.data() ); //NEW Saving the convoluted matrix
                
                //err = nc_put_vara_double( ncid_out, id2dX.at(record_name+"_Xgrid"),
                //startX2d_out, count2d_X, transferH2dX.data() );  //NEW saving the data on the X-grid 
                          
                //err = nc_put_vara_double( ncid_out, id2dX.at(record_name+"_conv1d_LCFS"),
                //start1d_out, count1d_conv, conv1d.data() );  //NEW saving the 1d convoluted vector
                
                //err = nc_put_vara_double( ncid_out, id2dX.at(record_name+"_partX2d"),
                //startX2d_out, count2d_X, part_transferH2dX.data() ); 		 //NEW saving the Cutted grid for partial fsa in the X-grid 
                
                err = nc_put_vara_double( ncid_out, id2dX["xcc"], startX2d_out, count2d_X, gridX2d.map()[0].data());//NEW SAving the maps and metrics of the new coordinates 
                err = nc_put_vara_double( ncid_out, id2dX["ycc"], startX2d_out, count2d_X, gridX2d.map()[1].data());
                err = nc_put_vara_double( ncid_out, id2dX["volX2d_sqrt(g)"], startX2d_out, count2d_X, volX2d.data());
				err = nc_put_vara_double( ncid_out, id2dX["conv_volX2d_sqrt(g)"], startX2d_out, count2d_X, conv_volX2d.data());
				err = nc_put_vara_double( ncid_out, id2dX["eta_coord"], start1d_out, count1d_conv, eta_c.data());
                
				}
                
                    
                    
                //4. Read 2d variable and compute fluctuations
                available = true;
                try{
                    err = nc_inq_varid(ncid, (record.name+"_2d").data(), &dataID);
                } catch ( file::NC_Error& error)
                {
                    if(  i == 0)
                    {
                        std::cerr << error.what() <<std::endl;
                        std::cerr << "Offending variable is "<<record.name+"_2d\n";
                        std::cerr << "Writing zeros ... \n";
                    }
                    available = false;
                }
                if( available)
                {
                    err = nc_get_vara_double( ncid, dataID, start2d, count2d,
                        t2d_mp.data());  //HERE t2d_mp IS THE DATA 
                    if( record_name[0] == 'j')
                        dg::blas1::pointwiseDot( t2d_mp, dvdpsip2d, t2d_mp ); //HERE WE SIMPLY MULTIPLY BY THE DVDPSIP2D
                    dg::blas1::axpby( 1.0, t2d_mp, -1.0, transferH2d); //HERE WE SUBSTRACT THE AVERAGE TO GET THE FLUCTUATIONS AND SAVE IT IN TRANSFER H2D
                    err = nc_put_vara_double( ncid_out, id2d.at(record_name+"_fluc2d"),
                        start2d_out, count2d, transferH2d.data() );

                    //5. flux surface integral/derivative
                    double result =0.;
                    if( record_name[0] == 'j') //j indicates a flux
                    {
                        dg::blas2::symv( dpsi, fsa1d, t1d);
                        dg::blas1::pointwiseDivide( t1d, dvdpsip, transfer1d);

                        result = dg::interpolate( dg::xspace, fsa1d, 0., g1d_out);
                    }
                    else
                    {
                        dg::blas1::pointwiseDot( fsa1d, dvdpsip, t1d);
                        transfer1d = dg::integrate( t1d, g1d_out);

                        result = dg::interpolate( dg::xspace, transfer1d, 0., g1d_out);
                    }
                    err = nc_put_vara_double( ncid_out, id1d.at(record_name+"_ifs"),
                        start1d_out, count1d, transfer1d.data());
                    //flux surface integral/derivative on last closed flux surface
                    err = nc_put_vara_double( ncid_out, id0d.at(record_name+"_ifs_lcfs"),
                        start2d_out, count2d, &result );
                    //6. Compute norm of time-integral terms to get relative importance
                    if( record_name[0] == 'j') //j indicates a flux
                    {
                        dg::blas2::symv( dpsi, fsa1d, t1d);
                        dg::blas1::pointwiseDivide( t1d, dvdpsip, t1d); //dvjv
                        dg::blas1::pointwiseDot( t1d, t1d, t1d);//dvjv2
                        dg::blas1::pointwiseDot( t1d, dvdpsip, t1d);//dvjv2
                        transfer1d = dg::integrate( t1d, g1d_out);
                        result = dg::interpolate( dg::xspace, transfer1d, 0., g1d_out);
                        result = sqrt(result);
                    }
                    else
                    {
                        dg::blas1::pointwiseDot( fsa1d, fsa1d, t1d);
                        dg::blas1::pointwiseDot( t1d, dvdpsip, t1d);
                        transfer1d = dg::integrate( t1d, g1d_out);

                        result = dg::interpolate( dg::xspace, transfer1d, 0., g1d_out);
                        result = sqrt(result);
                    }
                    err = nc_put_vara_double( ncid_out, id0d.at(record_name+"_ifs_norm"),
                        start2d_out, count2d, &result );
                }
                else
                {
                    dg::blas1::scal( transferH2d, 0.);
                    dg::blas1::scal( transfer1d, 0.);
                    double result = 0.;
                    err = nc_put_vara_double( ncid_out, id2d.at(record_name+"_fluc2d"),
                        start2d_out, count2d, transferH2d.data() );
                    err = nc_put_vara_double( ncid_out, id1d.at(record_name+"_ifs"),
                        start1d_out, count1d, transfer1d.data());
                    err = nc_put_vara_double( ncid_out, id0d.at(record_name+"_ifs_lcfs"),
                        start2d_out, count2d, &result );
                    err = nc_put_vara_double( ncid_out, id0d.at(record_name+"_ifs_norm"),
                        start2d_out, count2d, &result );
                }

            }


        } //end timestepping
        err = nc_close(ncid);
    }
    err = nc_close(ncid_out);

    return 0;
}
